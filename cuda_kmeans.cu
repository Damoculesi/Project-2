#include "hip/hip_runtime.h"
#include "kmeans.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <cmath>
#include <limits>
#include <cfloat>
#include <chrono>

// Kernel to assign data points to the nearest centroid
__global__ void assignPointsToCentroids(const double* data_points, const double* centroids, int* cluster_assignments, int num_points, int num_clusters, int dims) {
    int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (point_idx >= num_points) return;

    double min_distance = DBL_MAX;
    int closest_cluster = -1;

    for (int c = 0; c < num_clusters; ++c) {
        double distance = 0.0;
        for (int d = 0; d < dims; ++d) {
            double diff = data_points[point_idx * dims + d] - centroids[c * dims + d];
            distance += diff * diff;
        }
        if (distance < min_distance) {
            min_distance = distance;
            closest_cluster = c;
        }
    }

    cluster_assignments[point_idx] = closest_cluster;
}

// Kernel to compute sums and counts for centroid update
__global__ void updateCentroids(const double* data_points, const int* cluster_assignments, double* centroid_sums, int* points_per_cluster, int num_points, int num_clusters, int dims) {
    int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (point_idx >= num_points) return;

    int cluster_id = cluster_assignments[point_idx];

    for (int d = 0; d < dims; ++d) {
        atomicAdd(&centroid_sums[cluster_id * dims + d], data_points[point_idx * dims + d]);
    }

    atomicAdd(&points_per_cluster[cluster_id], 1);
}

// Kernel to compute new centroids by averaging and update centroids
__global__ void computeNewCentroids(const double* old_centroids, double* centroids, const double* centroid_sums, const int* points_per_cluster, int num_clusters, int dims) {
    int cluster_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (cluster_id >= num_clusters) return;

    if (points_per_cluster[cluster_id] > 0) {
        for (int d = 0; d < dims; ++d) {
            double new_value = centroid_sums[cluster_id * dims + d] / points_per_cluster[cluster_id];
            centroids[cluster_id * dims + d] = new_value;
        }
    } else {
        // Handle empty clusters by keeping the old centroid
        for (int d = 0; d < dims; ++d) {
            centroids[cluster_id * dims + d] = old_centroids[cluster_id * dims + d];
        }
    }
}

// Main function to run K-Means clustering using CUDA
std::pair<int, double> runKMeansCUDA(const std::vector<std::vector<double>>& data_points,
                                     std::vector<std::vector<double>>& centroids,
                                     int max_iters, double threshold) {

    int num_points = data_points.size();
    int num_clusters = centroids.size();
    int dims = centroids[0].size();

    // Flatten data_points and centroids for GPU
    std::vector<double> flat_data_points(num_points * dims);
    std::vector<double> flat_centroids(num_clusters * dims);
    for (int i = 0; i < num_points; ++i) {
        for (int d = 0; d < dims; ++d) {
            flat_data_points[i * dims + d] = data_points[i][d];
        }
    }
    for (int c = 0; c < num_clusters; ++c) {
        for (int d = 0; d < dims; ++d) {
            flat_centroids[c * dims + d] = centroids[c][d];
        }
    }

    // Allocate memory on GPU
    double* d_data_points;
    double* d_centroids;
    double* d_old_centroids;  // For convergence check
    int* d_cluster_assignments;
    hipMalloc(&d_data_points, num_points * dims * sizeof(double));
    hipMalloc(&d_centroids, num_clusters * dims * sizeof(double));
    hipMalloc(&d_old_centroids, num_clusters * dims * sizeof(double));
    hipMalloc(&d_cluster_assignments, num_points * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_data_points, flat_data_points.data(), num_points * dims * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, flat_centroids.data(), num_clusters * dims * sizeof(double), hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = (num_points + block_size - 1) / block_size;

    int iterations = 0;

    // Allocate memory for centroid sums and point counts
    double* d_centroid_sums;
    int* d_points_per_cluster;
    hipMalloc(&d_centroid_sums, num_clusters * dims * sizeof(double));
    hipMalloc(&d_points_per_cluster, num_clusters * sizeof(int));

    // Prepare for convergence check
    std::vector<double> flat_old_centroids = flat_centroids;

    // Start timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    while (iterations < max_iters) {
        // Step 1: Assign points to nearest centroid
        assignPointsToCentroids<<<grid_size, block_size>>>(d_data_points, d_centroids, d_cluster_assignments, num_points, num_clusters, dims);
        hipDeviceSynchronize();

        // Step 2: Reset centroid sums and point counts
        hipMemset(d_centroid_sums, 0, num_clusters * dims * sizeof(double));
        hipMemset(d_points_per_cluster, 0, num_clusters * sizeof(int));

        // Step 3: Update centroid sums and counts
        updateCentroids<<<grid_size, block_size>>>(d_data_points, d_cluster_assignments, d_centroid_sums, d_points_per_cluster, num_points, num_clusters, dims);
        hipDeviceSynchronize();

        // Step 4: Copy centroids to d_old_centroids for convergence check
        hipMemcpy(d_old_centroids, d_centroids, num_clusters * dims * sizeof(double), hipMemcpyDeviceToDevice);

        // Step 5: Compute new centroids and update centroids on device
        int centroid_block_size = 256;
        int centroid_grid_size = (num_clusters + centroid_block_size - 1) / centroid_block_size;
        computeNewCentroids<<<centroid_grid_size, centroid_block_size>>>(d_old_centroids, d_centroids, d_centroid_sums, d_points_per_cluster, num_clusters, dims);
        hipDeviceSynchronize();

        // Step 6: Copy centroids back to host to check for convergence
        hipMemcpy(flat_centroids.data(), d_centroids, num_clusters * dims * sizeof(double), hipMemcpyDeviceToHost);

        // Step 7: Check for convergence using Euclidean distance
        bool converged = true;
        for (int c = 0; c < num_clusters; ++c) {
            double dist = 0.0;
            for (int d = 0; d < dims; ++d) {
                double diff = flat_centroids[c * dims + d] - flat_old_centroids[c * dims + d];
                dist += diff * diff;
            }
            double threshold_squared = threshold * threshold;
            if (dist > threshold_squared) {
                converged = false;
                break;  // Exit early if any centroid has not converged
            }
        }

        // Prepare for next iteration
        flat_old_centroids = flat_centroids;

        if (converged) {
            break;
        }

        iterations++;
    }

    // End timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed_time = 0;
    hipEventElapsedTime(&elapsed_time, start, stop);
    double total_time = static_cast<double>(elapsed_time);

    // Copy final centroids back to host
    for (int c = 0; c < num_clusters; ++c) {
        for (int d = 0; d < dims; ++d) {
            centroids[c][d] = flat_centroids[c * dims + d];
        }
    }

    // Free GPU memory
    hipFree(d_data_points);
    hipFree(d_centroids);
    hipFree(d_old_centroids);
    hipFree(d_cluster_assignments);
    hipFree(d_centroid_sums);
    hipFree(d_points_per_cluster);

    return std::make_pair(iterations, total_time);
}
